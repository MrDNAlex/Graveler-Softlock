#include "hip/hip_runtime.h"
#include "BattleSimulation.cuh"

__global__ void SimulateBattle(int* turns, int* possibilities, int* iterations, int* paralysisCounts, unsigned long long* rngSeed)
{
	//Calculate GPU Core Index
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	//Dereference Variables
	int numOfTurns = *turns;
	int numOfPossibilities = *possibilities;
	int numOfIterations = *iterations;
	unsigned long long seed = *rngSeed;

	//Skip if GPU core Index is greater than the number of iterations
	if (index >= numOfIterations)
		return;

	//Initialize Move Count
	int moveCounts[4] = { 0, 0, 0, 0 };

	//Initialize Shared Memory for Paralysis Counts in a Block
	__shared__ int counts[1024];

	//Create a Shared Random Number Generator between all threads
	__shared__ hiprandState sharedRNG;
	if (threadIdx.x == 0)
		hiprand_init(seed, index, 0, &sharedRNG);
	__syncthreads();

	//Loop through the number of turns
	for (int i = 0; i < numOfTurns; i = i + 16)
	{
		//Generate a Random Number
		unsigned int paralysisOdd = hiprand(&sharedRNG);

		//Extract 2 Bits from the Random Number
		unsigned char random1 = (paralysisOdd >> 0) & 0x03;   // First 2 bits (0-3)
		unsigned char random2 = (paralysisOdd >> 2) & 0x03;   // Next 2 bits (0-3)
		unsigned char random3 = (paralysisOdd >> 4) & 0x03;   // Next 2 bits (0-3)
		unsigned char random4 = (paralysisOdd >> 6) & 0x03;   // Next 2 bits (0-3)
		unsigned char random5 = (paralysisOdd >> 8) & 0x03;   // Next 2 bits (0-3)
		unsigned char random6 = (paralysisOdd >> 10) & 0x03;  // Next 2 bits (0-3)
		unsigned char random7 = (paralysisOdd >> 12) & 0x03;  // Next 2 bits (0-3)
		unsigned char random8 = (paralysisOdd >> 14) & 0x03;  // Next 2 bits (0-3)
		unsigned char random9 = (paralysisOdd >> 16) & 0x03;  // Next 2 bits (0-3)
		unsigned char random10 = (paralysisOdd >> 18) & 0x03; // Next 2 bits (0-3)
		unsigned char random11 = (paralysisOdd >> 20) & 0x03; // Next 2 bits (0-3)
		unsigned char random12 = (paralysisOdd >> 22) & 0x03; // Next 2 bits (0-3)
		unsigned char random13 = (paralysisOdd >> 24) & 0x03; // Next 2 bits (0-3)
		unsigned char random14 = (paralysisOdd >> 26) & 0x03; // Next 2 bits (0-3)
		unsigned char random15 = (paralysisOdd >> 28) & 0x03; // Next 2 bits (0-3)
		unsigned char random16 = (paralysisOdd >> 30) & 0x03; // Last 2 bits (0-3)

		//Add to the right Array Index for the Move used
		moveCounts[random1]++;
		moveCounts[random2]++;
		moveCounts[random3]++;
		moveCounts[random4]++;
		moveCounts[random5]++;
		moveCounts[random6]++;
		moveCounts[random7]++;
		moveCounts[random8]++;
		moveCounts[random9]++;
		moveCounts[random10]++;
		moveCounts[random11]++;
		moveCounts[random12]++;
		moveCounts[random13]++;
		moveCounts[random14]++;
		moveCounts[random15]++;
		moveCounts[random16]++;
	}

	//Add the Paralysis Count to the Shared Memory
	counts[threadIdx.x] = moveCounts[0];

	//Find the Maximum Paralysis Count in the Thread Group
	int threadMax = 0;
	if (threadIdx.x == 0)
	{
		int max = 0;
		for (int i = 0; i < 1024; i++)
		{
			if (counts[i] > max)
				max = counts[i];
		}

		threadMax = max;
	}

	//Synchronize the Threads
	__syncthreads();

	//Check and replace the Global Paralysis Count if it's the highest so far
	if (threadIdx.x == 0)
	{
		atomicMax(paralysisCounts, threadMax);
	}
}

int SimulateBattles(int iterations, int turns, int possibilities, unsigned long long rngSeed)
{
	//Initialize GPU Variables
	int* gpuTurns = 0;
	int* gpuMoveRolls = 0;
	int* gpuPossibilities = 0;
	int* gpuInterations = 0;
	unsigned long long* gpuRNGSeed = 0;
	hiprandState gpuRNG;

	//Initialize CUDA Status
	hipError_t cudaStatus;

	//Get the GPU Device
	cudaStatus = hipSetDevice(0);

	//Assign Variables and Memory Space to the GPU
	cudaStatus = AssignVariable((void**)&gpuTurns, &turns, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuPossibilities, &possibilities, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuInterations, &iterations, sizeof(int));
	cudaStatus = AssignVariable((void**)&gpuRNGSeed, &rngSeed, sizeof(unsigned long long));
	//cudaStatus = AssignVariable((void**)&gpuRNG, &state, sizeof(hiprandState));
	cudaStatus = AssignMemory((void**)&gpuMoveRolls, sizeof(int));

	//Calculate the number of blocks and threads
    int threads = 1024;
	int blocks = (iterations + threads - 1) / threads;

	//Run the Simulation on the GPU
	SimulateBattle << <blocks, threads >> > (gpuTurns, gpuPossibilities, gpuInterations, gpuMoveRolls, gpuRNGSeed);

	//Synchronize the GPU (Wait for calculations to finish)
	cudaStatus = hipDeviceSynchronize();

	//Initialize the Move Rolls Array
	int* moveRolls = new int[1];
	
	//Retreive the Move Rolls from the GPU
	cudaStatus = GetVariable(moveRolls, gpuMoveRolls, sizeof(int));

	//Free Up the GPU Memory
	hipFree(gpuTurns);
	hipFree(gpuPossibilities);
	hipFree(gpuMoveRolls);
	hipFree(gpuInterations);
	hipFree(gpuRNGSeed);

	return moveRolls[0];
}